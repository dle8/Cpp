
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void kernel_A( float *g_data, int dimx, int dimy, int niterations)
{
	//change global memory access so that we have coalesced access

	int ix = threadIdx.x;
	int iy  = blockIdx.y*blockDim.y + threadIdx.y;
	int index = blockIdx.x*blockDim.x + ix;
	int idx = iy*dimx + index;

	float value = g_data[idx];
	
	if(ix & 1){

	    for(int i=0; i<niterations; i++)
	    {
	    	value += sqrtf( logf(value) + 1.f );
		}
	}

	else{

		for(int i=0; i<niterations; i++)
		{
			value += sqrtf( cosf(value) + 1.f );
		}
	}

	g_data[idx] = value;
}

float timing_experiment( void (*kernel)( float*, int,int,int), float *d_data, int dimx, int dimy, int niterations, int nreps, int blockx, int blocky )
{
	float elapsed_time_ms=0.0f;
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop  );

	dim3 block( blockx, blocky);
	dim3 grid( dimx/block.x, dimy/block.y );

	hipEventRecord( start, 0 );
	for(int i=0; i<nreps; i++)	// do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
		kernel<<<grid,block>>>( d_data, dimx,dimy, niterations);
	hipEventRecord( stop, 0 );
	hipDeviceSynchronize();
	hipEventElapsedTime( &elapsed_time_ms, start, stop );
	elapsed_time_ms /= nreps;

	hipEventDestroy( start );
	hipEventDestroy( stop );

	return elapsed_time_ms;
}

int main()
{
	int dimx = 2*1024;
	int dimy = 2*1024;

	int nreps = 10;
	int niterations = 20;

	int nbytes = dimx*dimy*sizeof(float);

	float *d_data=0, *h_data=0;
	hipMalloc( (void**)&d_data, nbytes );
	if( 0 == d_data )
	{
		printf("couldn't allocate GPU memory\n");
		return -1;
	}
	printf("allocated %.2f MB on GPU\n", nbytes/(1024.f*1024.f) );
	h_data = (float*)malloc( nbytes );
	if( 0 == h_data )
	{
		printf("couldn't allocate CPU memory\n");
		return -2;
	}
	printf("allocated %.2f MB on CPU\n", nbytes/(1024.f*1024.f) );
	for(int i=0; i<dimx*dimy; i++)
		h_data[i] = 10.f + rand() % 256;
	hipMemcpy( d_data, h_data, nbytes, hipMemcpyHostToDevice );

	float elapsed_time_ms=0.0f;

	elapsed_time_ms = timing_experiment( kernel_A, d_data, dimx,dimy, niterations, nreps, 32, 32);
	printf("A:  %8.2f ms\n", elapsed_time_ms );

	printf("CUDA: %s\n", hipGetErrorString( hipGetLastError() ) );

	if( d_data )
		hipFree( d_data );
	if( h_data )
		free( h_data );

	hipDeviceReset();

	return 0;
}


