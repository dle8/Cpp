
#include <hip/hip_runtime.h>
__global__ void sequential_reduction_kernel(float* g_out, float *g_in, unsigned int size) {
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s_data[];
    // Maximizing memory bandwidth with grid-strided loops. Sequential threads load consecutive locations in global mem array g_in
    float input = 0.f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x) input += g_in[i];
    s_data[threadIdx.x] = input;
    __syncthreads();

    // do reduction - sequential addressing to avoid thread divergence in reduction_shared_kernel.cu
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        g_out[blockIdx.x] = s_data[0];
    }
}