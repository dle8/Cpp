
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

const int NUM_BINS = 7;
const int NUM_BLOCKS = 16;

void init(int* &a, int sz) {
    a = (int*) calloc(sz, sizeof(int));
}

void print(int* a, int sz) {
    for (int i = 0; i < sz; ++i) {
        cout << a[i] << " ";
    }
    cout << '\n';
}

__global__ void histogram(char* buffer, int sz, int* bins) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // All threads handle blockDim.x * gridDim.x consecutive elements in each iteration
    for (int i = idx; i < sz; i += blockDim.x * gridDim.x) {
        int alphabet_pos = buffer[i] - 'a';
        if (alphabet_pos >= 0 && alphabet_pos < 26) atomicAdd(&bins[alphabet_pos / 4], 1);
    }
}

int main() {
    string hstr;
    cin >> hstr;
    int * hbins, *dbins;
    char* dstr;
    init(hbins, NUM_BINS);
    hipMalloc(&dbins, NUM_BINS * sizeof(int));
    hipMalloc(&dstr, hstr.size() * sizeof(char));
    hipMemcpy(dstr, hstr.c_str(), hstr.size() * sizeof(char), hipMemcpyHostToDevice);

    dim3 block(NUM_BLOCKS);
    dim3 grid((hstr.size() - 1) / NUM_BLOCKS + 1);

    histogram<<<grid, block>>>(dstr, hstr.size(), dbins);

    hipMemcpy(hbins, dbins, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);
    print(hbins, NUM_BINS);

    free(hbins);
    hipFree(dbins);
    return 0;
}